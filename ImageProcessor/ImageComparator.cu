#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <time.h>

// GPU�Ōv�Z����ۂ̊֐�
// __global__�̓z�X�g����Ă΂�ăf�o�C�X���Ŏ��s�����
// �߂�l��void�̂݁������Ɍ��ʂ�����ϐ���n���K�v������@������Ăяo�������猩��Ȃ��ƈӖ����Ȃ�����|�C���^�ł���K�v������
// C#��byte�^�����Ȃ�����1bit��C++����unsigned char�ɂȂ�炵��
// ���Ȃ݂�char��signed char��unsigned char�̂ǂ��炩�ɃR���p�C�����U�蕪���邩�璆�g���ǂ��炩�͎����I�Ɍ��܂�炵��
// unsigned char�͕����Ȃ���0�`255�@signed char�͕����t����-128 �` 127
// ���Ȃ݂�signed char��C#����sbyte�ɂȂ�Ƃ��@��������Java���Ƃ��ꂪbyte�炵���@��₱����
__global__ void gpu_function(int* result, unsigned char* b1, unsigned char* b2)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	result[i] = b1[i] + b2[i];
}

// main function
int main(unsigned char* b1Ptr, unsigned char* b2Ptr, int arrayLength)
{
	//�O���b�h�̒��Ƀu���b�N�������Ă��̒��ɃX���b�h������
	// 1�u���b�N���̃X���b�h�� �ő吔��512�炵��
	int threadNum = 512;

	for (int i = 0; i < arrayLength; i++)
	{
		int x = b1Ptr[i];
		std::cout << i << std::endl;
	}

	// �z��̃T�C�Y��z��v�f�̃T�C�Y�Ŋ���Ɣz��̗v�f����������炵��
	// 32bit��int�^��10�������Ƃ�����z��̑傫����320�@�v�f�̃T�C�Y��32�@�����10�ŗv�f�����킩��A��
	//int inputSize = sizeof(b1) / sizeof(b1[0]);
	//int editedSize;

	//std::cout << "in_" << inputSize << std::endl;

	// �����X���b�h���̔{������Ȃ��Ƃ��͒�������
	/*if (inputSize % threadNum != 0)
	{
		editedSize = inputSize % threadNum;
	}

	std::cout << "ed_" << editedSize << std::endl;*/

	//unsigned char* e1 = new unsigned char[editedSize];
	//unsigned char* e2 = new unsigned char[editedSize];

	// �傫���𒲐�������̔z��ɗv�f���R�s�[����
	//for (int i = 0; i <inputSize ; i++)
	//{
	//	e1[i] = b1[i];
	//	e2[i] = b2[i];
	//}

	//// �󂢂Ă��镔����0�Ŗ��߂�
	//for (int i = 0; i < editedSize - inputSize; i++)
	//{
	//	// 0�`inputSize-1�܂ł͊��ɑ���ς݁@�c��̕����ɓ���Ă���
	//	e1[i + inputSize -1] = 0;
	//	e2[i + inputSize -1] = 0;
	//}

	//// ���ʂ����邽�߂̕ϐ�
	//// �z��̑傫���͒萔����Ȃ��ƃ_���炵��
	//int* result = new int[editedSize];

	//// �f�o�C�X���̔z���p��
	//int* d_result = new int[editedSize];
	//unsigned char* d_e1 = new unsigned char[editedSize];
	//unsigned char* d_e2 = new unsigned char[editedSize];

	//// �f�o�C�X�̃��������m��
	//hipMalloc(&d_result, editedSize * sizeof(int));
	//hipMalloc(&d_e1, editedSize * sizeof(unsigned char));
	//hipMalloc(&d_e2, editedSize * sizeof(unsigned char));

	//// �z�X�g���̔z����f�o�C�X���ɃR�s�[
	//hipMemcpy(d_result, result, editedSize * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(d_e1, e1, editedSize * sizeof(unsigned char), hipMemcpyHostToDevice);
	//hipMemcpy(d_e2, e2, editedSize * sizeof(unsigned char), hipMemcpyHostToDevice);

	//// �u���b�N�̑傫�������߂�
	//dim3 block(inputSize / threadNum, 1);
	//// �O���b�h�̑傫�������߂�
	//dim3 grid(1, 1);

	//gpu_function <<<grid, block>>> (d_result, d_e1, d_e2);

	//int r;
	//for (int i = 0; i < editedSize; i++)
	//{
	//	r += result[i];
	//}

	// new������K��delete����A��
	/*delete[] e1;
	delete[] e2;*/
	/*delete[] d_e1;
	delete[] d_e2;
	delete[] result;
	delete[] d_result;*/

	return 0;
	//return r;
}